#include "hip/hip_runtime.h"
#include <cstdio>
#include <GkylWavePropagation.h>
#include <GkylEuler.h>

__device__ static void calcDelta(
  const double *ql, const double *qr, double *delta, const int meqn)
{
  for (int i = 0; i < meqn; i++) {
    delta[i] = qr[i] - ql[i];
  }
}

__device__ static void calcFirstOrderGud(
  const double dtdx, double *ql, double *qr, const double *amdq,
  const double *apdq, const int meqn)
{
    /* if (threadIdx.x ==0 && blockIdx.x==0) */
    /*   printf("ql[0] before %13g\n", ql[0]); */
  for (int i = 0; i< meqn; i++) {
    /* qr[i] -= dtdx * apdq[i]; */
    /* ql[i] -= dtdx * amdq[i]; */
    // XXX calling __threadfence_system() between two calcFirstOrderGud dummy
    // calls fails occasionally with small numThreads
    atomicAdd(qr+i, -dtdx * apdq[i]);
    atomicAdd(ql+i, -dtdx * amdq[i]);
  }
    /* if (threadIdx.x ==0 && blockIdx.x==0) */
    /*   printf("ql[0] after %13g\n", ql[0]); */
}

__device__ static double calcCfla(
  const double cfla, const double dtdx, const double *s, const int mwave)
{
  double c = cfla;
  for (int i = 0; i < mwave; i ++) {
    c = max(c, dtdx * abs(s[i]));
  }
  return c;
}

__device__ static double waveDotProd(
    const double *waves, const double *waves1, const int mw,
    const int meqn) {
  double result = 0.;
  for (int i = 0; i < meqn; i++) {
    result += waves[meqn*mw+i] * waves[meqn*mw+i];
  }
  return result;
}

__device__ static inline double limiter_minMod(const double r) {
   return max(0., min(1., r));
}

__device__ static void limitWaves(
    double *waveSlice, const double *speedSlice, const int mwave,
    const int meqn) {
  int i = threadIdx.x + 1;  // FIXME
  int jump = meqn * mwave;
  for (int mw = 0; mw < mwave; mw++ ){
    const double wnorm2 = waveDotProd(
        waveSlice+i*jump, waveSlice+i*jump, mw, meqn);
    double wlimitr = 1.;
    if (wnorm2 > 0) {
      double r;
        const double dotl = waveDotProd(
            waveSlice+(i-1)*jump, waveSlice+i*jump, mw, meqn);
        const double dotr = waveDotProd(
            waveSlice+(i+1)*jump, waveSlice+i*jump, mw, meqn);
        r = speedSlice[i] > 0 ? dotl/wnorm2 : dotr/wnorm2;
      wlimitr = limiter_minMod(r);
        printf("[%2d] dotl %13g dotr %13g wnorm2 %13g\n",
          i, dotl, dotr, wnorm2);
    }
    __threadfence_block();
    for (int me = 0; me < meqn; me++) {
      waveSlice[i*jump+mw*meqn] *= wlimitr;
    }
  }
}

__device__ static void secondOrderFlux(
  const double dtdx, const double s, const double *wave, double *fs,
  const int meqn) {
  double sfact = 0.5 * abs(s) * (1 - abs(s) * dtdx);
  for (int i = 0; i < meqn; i++) {
    fs[i] += sfact * wave[i];
  }
}

__device__ static void secondOrderUpdate(
    const double dtdx, const double *fs, const double *fs1, double *q,
    const int meqn) {
  for (int i = 0; i < meqn; i++) {
    q[i] -= dtdx * (fs1[i] - fs[i]);
  }
}

__device__ static void copyComponents(
    const double *ptrFrom, double *ptrTo, const int nComponents) {
  for (int i = 0; i < nComponents; i++) {
    ptrTo[i] = ptrFrom[i];
  }
}

__global__ void cuda_WavePropagation(
  GkylWavePropagation_t *hyper, GkylCartField_t *qIn, GkylCartField_t *qOut)
{

  GkylRange_t *localRange = qIn->localRange;
  GkylRange_t *localExtRange = qIn->localExtRange;
  GkylRange_t *localEdgeRange = qIn->localEdgeRange;
  GkylRange_t *localExtEdgeRange = qIn->localExtEdgeRange;
  int ndim = localRange->ndim;

  // set up indexers for localRange and qIn (localExtRange)
  Gkyl::GenIndexer localIdxr(localRange);
  Gkyl::GenIndexer extIdxr(localExtRange);
  Gkyl::GenIndexer edgeIdxr(localExtEdgeRange);
  Gkyl::GenIndexer fIdxr = qIn->genIndexer();

  // get setup data from GkylWavePropagation_t structure
  GkylRectCart_t *grid = qIn->grid;
  int *updateDirs = hyper->updateDirs;
  int numUpdateDirs = hyper->numUpdateDirs;
  Gkyl::Euler *eq = hyper->equation;
  GkylCartField_t *dtByCell = hyper->dtByCell;

  const int meqn = eq->numEquations();
  const int mwave = eq->numWaves();

  // XXX use meqn and mwave
  double delta[5];
  double amdq[5];
  double apdq[5];

  // declaring this dummy array shared seems to alleviate register pressure and
  // improve performance a bit
  extern __shared__ double dummy[];
  int linearIdx = threadIdx.x + blockIdx.x*blockDim.x;

  // assign buffer space for different usages
  int base = 0;

  const int baseWaveSlice = base;
  base += (meqn * mwave) * blockDim.x;
  double *waveSlice = dummy + baseWaveSlice;

  const int baseSpeedSlice = base;
  base += (mwave) * blockDim.x;
  double *speedSlice = dummy + baseSpeedSlice;

  const int baseFluxSlice = base;
  base += (meqn) * blockDim.x;
  double *fluxSlice = dummy + baseFluxSlice;

  // find buffer addresses for each thread
  // FIXME shall waves and s be created on the fly and then copied into slices
  double *waves = waveSlice + (meqn * mwave) * threadIdx.x;
  double *s = speedSlice + (mwave) * threadIdx.x;
  double *flux = fluxSlice + (meqn) * threadIdx.x;

  int idxC[3];
  int idxL[3];
  int idxR[3];

  // get i,j,k... index idxC
  localIdxr.invIndex(linearIdx, idxC);
  //edgeIdxr.invIndex(linearIdx, idxC);
  // if ndim>1, linearIdxC!=linearIdx since linearIdxC jumps due to ghost cells
  const int linearIdxC = fIdxr.index(idxC);

  const double *dx = grid->dx;

  double cfl = hyper->_cfl;
  double cflm = hyper->_cflm;
  double cfla = 0; // actual CFL number used

  const double *qInC = qIn->getDataPtrAt(linearIdxC);
  double *qOutC = qOut->getDataPtrAt(linearIdxC);

  /* if(linearIdx < localRange->volume()) { */
  /*   for(int i = 0; i < meqn; i++) { */
  /*     qOutC[i] = qInC[i]; */
  /*   } */
  /*   if (threadIdx.x == 0 && blockIdx.x == 0) { */
  /*     for(int i = 0; i < meqn; i++) { */
  /*       qOutC[i] = qInC[i]; */
  /*     } */
  /*   } */
  /* } */
    /* __syncthreads(); */
    /* if(linearIdx < localExtEdgeRange->volume()) */
    /*   printf("[%2d] qOut [%2d] %13g\n", linearIdx, idxC[0], qOutC[0]); */

  for(int i=0; i<numUpdateDirs; i++) {
    int dir = updateDirs[i] - 1;
    const double dtdx = hyper->dt / dx[dir];

    for(int d=0; d<ndim; d++) {
      idxL[d] = idxC[d];
      idxR[d] = idxC[d];
    }
    /* idxL[dir] = idxC[dir] - 1; */
    /* idxR[dir] = idxC[dir]; */
    idxL[dir] = idxC[dir] - 2;
    idxR[dir] = idxC[dir] - 1;

    const int linearIdxL = fIdxr.index(idxL);
    const int linearIdxR = fIdxr.index(idxR);
    const double *qInL = qIn->getDataPtrAt(linearIdxL);
    const double *qInR = qIn->getDataPtrAt(linearIdxR);

    double *qOutL = qOut->getDataPtrAt(linearIdxL);
    double *qOutR = qOut->getDataPtrAt(linearIdxR);

    /* __syncthreads(); */
    /* if(linearIdx < localExtEdgeRange->volume()) */
    /*   printf("[%2d] before L [%2d] %13g, R [%2d] %13g; wave %13g; amdq %13g; qOutL %13g\n", */
    /*       linearIdx, idxL[dir], qInL[0], idxR[dir], qInR[0], waves[0], amdq[0], qOutL[0]); */

    if(linearIdx < localExtEdgeRange->volume()) {
      calcDelta(qInL, qInR, delta, meqn);

      eq->rp(dir, delta, qInL, qInR, waves, s);
      eq->qFluctuations(dir, qInL, qInR, waves, s, amdq, apdq);

      calcFirstOrderGud(dtdx, qOutL, qOutR, amdq, apdq, meqn);
      // XXX following fails with small numThreads
      /* calcFirstOrderGud(dtdx, qOutL, dummy, amdq, apdq, meqn); */
      /* __threadfence_system(); */
      /* calcFirstOrderGud(dtdx, dummy, qOutR, amdq, apdq, meqn); */

      cfla = calcCfla(cfla, dtdx, s, mwave);

    }

    /* __syncthreads(); */
    /* if(linearIdx < localExtEdgeRange->volume()) */
    /*   printf("[%2d] L [%2d] %13g, R [%2d] %13g; wave %13g; amdq %13g; qOutL %13g\n", */
    /*       linearIdx, idxL[dir], qInL[0], idxR[dir], qInR[0], waves[0], amdq[0], qOutL[0]); */

    if(linearIdx < localEdgeRange->volume()) {
      limitWaves(waveSlice, speedSlice, mwave, meqn);
    }

    __syncthreads();
    if(linearIdx < localExtEdgeRange->volume()) {
      printf("[%2d] limited wave %13g\n",
          linearIdx, waves[0]);
    }

    /* if(linearIdx < localEdgeRange->volume()) { */
    /*   int idx = threadIdx.x + 1; */
    /*   double *waves = waveSlice + (meqn * mwave) * i; */
    /*   double *s = speedSlice + (mwave) * i; */
    /*   double *flux = fluxSlice + (meqn) * i; */
    /*  */
    /*   for (int c = 0; c < meqn; c++) { */
    /*     flux[c] = 0; */
    /*   } */
    /*  */
    /*   for (int mw = 0; mw < mwave; mw++) { */
    /*     secondOrderFlux(dtdx, s[mw], waves+mw*meqn, flux, meqn); */
    /*   } */
    /* } */
    /*  */
    /* if(linearIdx < localRange->volume()) { */
    /*   int idx = threadIdx.x + 1; */
    /*   double *flux = fluxSlice + (meqn) * i; */
    /*   double *flux1 = fluxSlice + (meqn) * (i + 1); */
    /*   idxC[dir] += 1; */
    /*   const int linearIdxCC = fIdxr.index(idxC); */
    /*   double *qOutCC = qOut->getDataPtrAt(linearIdxCC); */
    /*   secondOrderUpdate(dtdx, flux, flux1, qOutCC, meqn); */
    /* } */
  }

  dtByCell->getDataPtrAt(linearIdxC)[0] = hyper->dt * cfl/cfla;
}

void wavePropagationAdvanceOnDevice(
  int numBlocks, int numThreads, GkylWavePropagation_t *hyper,
  GkylCartField_t *qIn, GkylCartField_t *qOut)
{
  Gkyl::Euler *eq = hyper->equation;
  // XXX
  const int meqn = 5; // eq->numEquations();
  const int mwave = 1; // eq->numWaves();
  const int nComponents = (mwave + mwave * meqn + meqn);
  const int sharedMemSize = numThreads * nComponents;

  hipFuncSetAttribute(reinterpret_cast<const void*>(
    cuda_WavePropagation), hipFuncAttributeMaxDynamicSharedMemorySize,
    sharedMemSize*sizeof(double));
  cuda_WavePropagation<<<numBlocks, numThreads, sharedMemSize*sizeof(double)>>>(
    hyper, qIn, qOut);
}

__global__ void setDtOnDevice(GkylWavePropagation_t *hyper, double dt) {
  hyper->dt = dt;
}

void setDt(GkylWavePropagation_t *hyper, double dt) {
  setDtOnDevice<<<1, 1>>>(hyper, dt);
}
